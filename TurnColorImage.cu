#include "hip/hip_runtime.h"
#include "TurnColorImage.cuh"


template < typename T > 
__device__ void swap(T &a, T &b) {
	T temp = a;
	a = b;
	b = temp;
}


__global__ void ReverceGPU(UINT * deviceImage, const int Height, const int Stride, int HeightOnTwo) {

	int i = blockIdx.x * 8 + threadIdx.x;
	int j = blockIdx.y * 32 + threadIdx.y;

	if (i >= Stride || j >= HeightOnTwo) {
		return;
	}
	swap(deviceImage[(Height - j) * Stride - Stride + i], deviceImage[j * Stride  + i]);


}

TurnColorImage::TurnColorImage() {
	GdiplusStartup(&GDIPlusToken, &GDIInput, NULL);
	bitMapCPU = new Bitmap(L"inputImage.jpg");
	bitMapGPU = new Bitmap(L"inputImage.jpg");
	bitmapDataCPU = new BitmapData;
	bitmapDataGPU = new BitmapData;
	bitMapCPU->LockBits(NULL, ImageLockMode::ImageLockModeWrite, PixelFormat24bppRGB, bitmapDataCPU);
	bitMapGPU->LockBits(NULL, ImageLockMode::ImageLockModeWrite, PixelFormat24bppRGB, bitmapDataGPU);
	pixCPU = (UINT*)bitmapDataCPU->Scan0;
	pixGPU = (UINT*)bitmapDataGPU->Scan0;
	hipMalloc((void**)&deviceImage, bitmapDataGPU->Height * bitmapDataGPU->Stride / 4 * sizeof(UINT));
	hipMemcpy(deviceImage, pixGPU, bitmapDataGPU->Height * bitmapDataGPU->Stride / 4 * sizeof(UINT), hipMemcpyHostToDevice);
}
TurnColorImage::~TurnColorImage(){
	delete bitmapDataCPU;
	delete bitmapDataGPU;
	delete bitMapCPU;
	delete bitMapGPU;
	GdiplusShutdown(GDIPlusToken);
}

void TurnColorImage::check() {
	bool compareFlag = false;
	for (auto i = 0; i < bitmapDataGPU->Height; i++) {
		for (auto j = 0; j < bitmapDataGPU->Stride / 4; j++) {
			if (pixCPU[i * bitmapDataGPU->Stride / 4 + j] != pixGPU[i * bitmapDataGPU->Stride / 4 + j]) {
				compareFlag = true;
				std::cout << i << " " << j << std::endl;
			}
		}
	}
	std::cout << ((compareFlag) ? "Images are not equals" : "Images are equals") << std::endl;
}


void TurnColorImage::CPUReverse() {

	auto startTick = __rdtsc();
	if (bitmapDataCPU->Height % 2 == 0) {
		for (int i = bitmapDataCPU->Height / 2 - 1, j = bitmapDataCPU->Height / 2; i >= 0; i--, j++) {
			for (auto k = 0; k < bitmapDataCPU->Stride / 4; k++) {
				std::swap(pixCPU[i * bitmapDataCPU->Stride / 4 + k],
					pixCPU[j * bitmapDataCPU->Stride / 4 + k]);
			}
		}
	}
	else {
		for (int i = bitmapDataCPU->Height / 2 - 1, j = bitmapDataCPU->Height / 2 + 1; i >= 0; i--, j++) {
			for (auto k = 0; k < bitmapDataCPU->Stride / 4; k++) {
				std::swap(pixCPU[i * bitmapDataCPU->Stride / 4 + k],
					pixCPU[j * bitmapDataCPU->Stride / 4 + k]);
			}
		}
	}
	auto finishTick = __rdtsc();
	std::cout << "Run Time CPU = " << double((finishTick - startTick) / CLK_TCK / 1000) << std::endl;
	bitMapCPU->UnlockBits(bitmapDataCPU);
	CLSID pngClsid;
	GetEncoderClsid(L"image/jpeg", &pngClsid);
	bitMapCPU->Save(L"CPU_out.jpg", &pngClsid, NULL);


}

void TurnColorImage::CUDAReverse() {
	hipEvent_t start, stop;
	float timer = 0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);


	dim3 threads(8, 32);
	dim3 blocks(ceil((bitmapDataGPU->Stride + threads.x  + 1) / 8), ceil((bitmapDataGPU->Height  + threads.x * 2 + 1) / 64));

	hipEventSynchronize(start);

	ReverceGPU <<< blocks, threads >>>  (deviceImage, bitmapDataGPU->Height, bitmapDataGPU->Stride / 4, bitmapDataGPU->Height / 2);

	hipError_t error = hipGetLastError();
	if (error != hipSuccess) {
		std::cout << hipGetErrorString(error) << std::endl;
	}
	hipEventRecord(stop);
	hipEventSynchronize(stop);


	hipMemcpy(pixGPU, deviceImage, bitmapDataGPU->Height * bitmapDataGPU->Stride / 4 * sizeof(UINT), hipMemcpyDeviceToHost);
	hipEventElapsedTime(&timer, start, stop);
	std::cout << "Run Time GPU = " << timer << std::endl;
	bitMapGPU->UnlockBits(bitmapDataGPU);
	CLSID pngClsid;
	GetEncoderClsid(L"image/jpeg", &pngClsid);
	bitMapGPU->Save(L"GPU_out.jpg", &pngClsid, NULL);
	check();
}

int TurnColorImage::GetEncoderClsid(const WCHAR* format, CLSID* pClsid)
{
	UINT  num = 0;          // number of image encoders
	UINT  size = 0;         // size of the image encoder array in bytes

	ImageCodecInfo* pImageCodecInfo = NULL;

	GetImageEncodersSize(&num, &size);
	if (size == 0)
		return -1;  // Failure

	pImageCodecInfo = (ImageCodecInfo*)(malloc(size));
	if (pImageCodecInfo == NULL)
		return -1;  // Failure

	GetImageEncoders(num, size, pImageCodecInfo);

	for (UINT j = 0; j < num; ++j)
	{
		if (wcscmp(pImageCodecInfo[j].MimeType, format) == 0)
		{
			*pClsid = pImageCodecInfo[j].Clsid;
			free(pImageCodecInfo);
			return j;  // Success
		}
	}

	free(pImageCodecInfo);
	return -1;  // Failure
}
